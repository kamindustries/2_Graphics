#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
// OpenGL Graphics includes
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
#include <timer.h>               // timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <hip/hip_vector_types.h>
#include "lodepng.h"
#include "writePNG.h"


// static void HandleError( hipError_t err, const char *file,  int line ) {
//     if (err != hipSuccess) {
//             printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
//             exit( EXIT_FAILURE );
//     }
// }
// #define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define MAX(a,b) ((a > b) ? a : b)
#define     DIM    256
#define     DT    .1

GLuint  bufferObj, bufferObj2;
GLuint  textureID;
// hipGraphicsResource_t resource[2];
hipGraphicsResource_t resource1;
hipGraphicsResource_t resource2;
float ttime = 0.0f;
float avgFPS = 0.0f;
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int frameNum = 0;
unsigned int frameCount = 0;
float4 *chemA, *chemB, *displayPtr;
bool runOnce = true;

// diffusion constants
float dA = 0.0002;
float dB = 0.00001;
float F = 0.05;
float k = 0.0675;

StopWatchInterface *timer = NULL;

// mouse controls
int mouse_old_x, mouse_old_y;
bool togSimulate = false;
int max_simulate = 0;
// int pause = 17500;

bool writeCpy = false;
bool writeDone = false;


///////////////////////////////////////////////////////////////////////////////
// Functions
///////////////////////////////////////
// Write
///////////////////////////////////////
void write(const char* _filename, float4* _img) {
    FILE* file;
    file = fopen(_filename, "wb");

    int totalCells = DIM * DIM;
    // double* dataDouble = new double[totalCells * 3];
    for (int i = 0; i < totalCells; i++) {
      fprintf(file, "%f\n", _img[i].x);
      fprintf(file, "%f\n", _img[i].y);
      fprintf(file, "%f\n", _img[i].z);
    }

    fclose(file);

    writeCpy = false;
    printf("Wrote file!\n");
}

void encodeOneStep(const char* _filename, const unsigned char* image, unsigned width, unsigned height) {
  /*Encode the image*/
  unsigned error = lodepng_encode32_file(_filename, image, width, height);
  /*if there's an error, display it*/
  if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

///////////////////////////////////////
// Compute FPS
///////////////////////////////////////
void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "CUDA || %3.1f fps", avgFPS);
    glutSetWindowTitle(fps);
}

///////////////////////////////////////
// Delete PBO
///////////////////////////////////////
void deletePBO(GLuint *pbo) 
{
    glDeleteBuffers(1, pbo);
    SDK_CHECK_ERROR_GL();
    *pbo = 0;
}

void deleteTexture(GLuint *tex)
{
    glDeleteTextures(1, tex);
    SDK_CHECK_ERROR_GL();
    *tex = 0;
}


///////////////////////////////////////
// CUDA Kernel
///////////////////////////////////////
__device__ int checkPosition(int _pos){
  int dmax = DIM*DIM;
  if (_pos < 0){
    // _pos = dmax+_pos;
    _pos += DIM;
    // return _pos % dmax;
    return _pos;
  }
  else return _pos % dmax;
}

__global__ void RunOnce( float4 *_chem) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  _chem[offset] = make_float4(0.,0.,0.,1.);
}

__global__ void DrawSquare( float4 *_chem ) {
  if (threadIdx.x > DIM || threadIdx.y > DIM) return;

  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + (y * blockDim.x * gridDim.x);

  // q1. draws a square
  float posX = (float)x/DIM;
  float posY = (float)y/DIM;
  // if ( x < 140 && x > 116 && y < 140 && y > 116 ) {
  if ( x < 200 && x > 116 && y < 140 && y > 30 ) {
  // if ( posX < .75 && posX > .45 && posY < .55 && posY > .45 ) {
  // if ( posX < m_x+.05 && posX > m_x-.05 && posY < m_y+.05 && posY > m_y-.05 ) {    //use mouse position
    _chem[offset] = make_float4(1.,1.,1.,1.);
  }

}

__global__ void Diffusion( float4 *_chem, float4 *_lap, float _difConst, int mouse_x, int mouse_y) {
  if (threadIdx.x > DIM || threadIdx.y > DIM) return;

  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int offset = x + (y * blockDim.x * gridDim.x);

  // constants
  // float xLength = (float)DIM/100.0;
  float xLength = 2.56;
  // float dx = (float)xLength/DIM;
  float dx = 0.01;
  float alpha = _difConst * DT / (dx*dx);

  // int n1 = (x+1)%DIM;
  // int n2 = (x-1)%DIM;
  // int n3 = (y+1)%DIM;
  // int n4 = (y-1)%DIM;

  // if (n2 < 0) n2 += DIM;
  // if (n4 < 0) n4 += DIM;

  // n1 = ((n1 + y * blockDim.x * gridDim.x)) % (DIM*DIM);
  // n2 = ((n2 + y * blockDim.x * gridDim.x)) % (DIM*DIM);
  // n3 = ((x + n3 * blockDim.x * gridDim.x)) % (DIM*DIM);
  // n4 = ((x + n4 * blockDim.x * gridDim.x)) % (DIM*DIM);
  // if (n1 > (y*DIM)+DIM) n1 -= DIM;

  int n1 = offset + 1;
  int n2 = offset - 1;
  int n3 = offset + DIM;
  int n4 = offset - DIM;

  if (n1 > ((DIM-1) + (y * blockDim.x * gridDim.x))) n1 -= DIM;
  if (n1 >= DIM*DIM) n1 -= DIM;

  if (n2 < (0 + (y * blockDim.x * gridDim.x))) n2 = ((DIM-1) + (y * blockDim.x * gridDim.x));
  if (n2 < 0) n2 += DIM;

  if (n3 >= DIM*DIM) n3 = x; 
  
  if (n4 < 0) n4 = (DIM*DIM) - DIM + x; 

  // int n1 = checkPosition((x+1) + y * blockDim.x * gridDim.x);
  // int n2 = checkPosition((x-1) + y * blockDim.x * gridDim.x);
  // int n3 = checkPosition(x + (y+1) * blockDim.x * gridDim.x);
  // int n4 = checkPosition(x + (y-1) * blockDim.x * gridDim.x);
  // __syncthreads();

  _lap[offset] = -4.0f * _chem[offset] + _chem[n1] + _chem[n2] + _chem[n3] + _chem[n4];
  _lap[offset] *= alpha;

}

__global__ void AddLaplacian( float4 *_chem, float4 *_lap) {
  if (threadIdx.x > DIM || threadIdx.y > DIM) return;

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  _chem[offset] += _lap[offset];
  _chem[offset].w = 1.0;

}

__global__ void React( float4 *_chemA, float4 *_chemB, float4 *_rA, float4 *_rB) {
  // if (threadIdx.x > DIM || threadIdx.y > DIM) return;

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  float F = 0.05;
  float k = 0.0675;
  float4 A = _chemA[offset];
  float4 B = _chemB[offset];

  float4 reactionA = make_float4(-A.x * (B.x*B.x) + (F * (1.0-A.x)),
                                -A.y * (B.y*B.y) + (F * (1.0-A.y)),
                                -A.z * (B.z*B.z) + (F * (1.0-A.z)),
                                -A.w * (B.w*B.w) + (F * (1.0-A.w))
                                );

  float4 reactionB = make_float4(A.x * (B.x*B.x) - (F+k)*B.x,
                                A.y * (B.y*B.y) - (F+k)*B.y,
                                A.z * (B.z*B.z) - (F+k)*B.z,
                                A.w * (B.w*B.w) - (F+k)*B.w
                                );

  _rA[offset] = reactionA * .1;
  _rB[offset] = reactionB * .1;

  // _chemA[offset] += (DT * reactionA); //need parenthesis
  // _chemA[offset].w = 1.0;

  // _chemB[offset] += (DT * reactionB);
  // _chemB[offset].w = 1.0;
}

__global__ void AddReaction( float4 *_chemA, float4 *_chemB, float4 *_rA, float4 *_rB) {
  // if (threadIdx.x > DIM || threadIdx.y > DIM) return;

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;

  _chemA[offset] += _rA[offset];
  _chemA[offset].w = 1.0;

  _chemB[offset] += _rB[offset];
  _chemB[offset].w = 1.0;
}

///////////////////////////////////////
// Simulate
///////////////////////////////////////
static void simulate( void ){

    for (int i = 0; i < 10; i++){
      float4 *laplacian;
      size_t  size;

      checkCudaErrors(hipMalloc((void**)&chemA, sizeof(float4)*DIM*DIM ));
      checkCudaErrors(hipMalloc((void**)&chemB, sizeof(float4)*DIM*DIM ));
      checkCudaErrors(hipMalloc((void**)&laplacian, sizeof(float4)*DIM*DIM ));
      
      float4 *rA, *rB;
      checkCudaErrors(hipMalloc((void**)&rA, sizeof(float4)*DIM*DIM ));
      checkCudaErrors(hipMalloc((void**)&rB, sizeof(float4)*DIM*DIM )); 

      dim3    grid(DIM/16,DIM/16);
      dim3    threads(16,16);
      // dim3    grid(12,12);
      // dim3    threads(16,16);

      // *!* important
      // load chem fields with color 0,0,0,1
      if (runOnce == true){
        RunOnce<<<grid,threads>>>(chemA);
        RunOnce<<<grid,threads>>>(chemB);
        runOnce = false;
      }

      DrawSquare<<<grid,threads>>>(chemB);

      if (frameNum > 0) {

        Diffusion<<<grid,threads>>>( chemA, laplacian, dA, mouse_old_x, mouse_old_y );
        AddLaplacian<<<grid,threads>>>( chemA, laplacian );

        RunOnce<<<grid,threads>>>(laplacian);

        Diffusion<<<grid,threads>>>( chemB, laplacian, dB, mouse_old_x, mouse_old_y );
        AddLaplacian<<<grid,threads>>>( chemB, laplacian );

        React<<<grid,threads>>>( chemA, chemB, rA, rB );
        AddReaction<<<grid,threads>>>( chemA, chemB, rA, rB );
      }

      hipGraphicsMapResources( 1, &resource1, 0 );
      checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&displayPtr, &size, resource1)); 
      checkCudaErrors(hipMemcpy(displayPtr, chemB, sizeof(float4)*DIM*DIM, hipMemcpyDeviceToHost ));
    
      // if (frameNum == 1000 || frameNum == 9000 || frameNum == 17000 ||
      //     frameNum == 19000 || frameNum == 21000 || frameNum == 23000) {
      if (frameNum % 500 == 0 && frameNum <= 10000) {
        writeCpy = true;
      }
      
      if (writeCpy) {
        float4* img_ptr = (float4*)malloc(sizeof(float4)*DIM*DIM);
        checkCudaErrors (hipMemcpy(img_ptr, chemB, sizeof(float4)*DIM*DIM, hipMemcpyDeviceToHost ));

        char filename_txt[1024 * sizeof(int) / 3 + 2];
        sprintf(filename_txt, "data/cuda_x%d.txt", frameNum);
        write(filename_txt, img_ptr);

        char filename_png[1024 * sizeof(int) / 3 + 2];
        sprintf(filename_png, "data/cuda_x%d.png", frameNum);
        writePNG(filename_png, img_ptr, 256, 256);
      }

      
      checkCudaErrors(hipGraphicsUnmapResources( 1, &resource1, 0 ));
      checkCudaErrors(hipFree(chemA));
      checkCudaErrors(hipFree(chemB));
      checkCudaErrors(hipFree(rA));
      checkCudaErrors(hipFree(rB));
      checkCudaErrors(hipFree(laplacian));
      
      frameNum++;
      // if (frameNum == pause) togSimulate = false;
    }


    // printf("chem b: %f", displayPtr[0].x);
    // printf("\r");
}


///////////////////////////////////////
// Draw
///////////////////////////////////////
static void draw_func( void ) {

  if (togSimulate) {
    simulate();
  }

  glClear(GL_COLOR_BUFFER_BIT);

  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
  glBindTexture(GL_TEXTURE_2D, textureID);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, DIM, DIM, GL_BGRA, GL_FLOAT, NULL);

  glBegin(GL_QUADS);
  glTexCoord2f( 0, 1.0f);
  glVertex3f(-1.0,1.0f,0);
  glTexCoord2f(0,0);
  glVertex3f(-1.0f,-1.0f,0);
  glTexCoord2f(1.0f,0);
  glVertex3f(1.0f,-1.0f,0);
  glTexCoord2f(1.0f,1.0f);
  glVertex3f(1.0f,1.0f,0);
  glEnd();

  glutSwapBuffers();

  computeFPS();
  ttime += 0.0001;
  glutPostRedisplay(); // causes draw to loop forever
  
  // printf("frame %d", frameNum);
  // printf("\r");

}

///////////////////////////////////////
// Close function
///////////////////////////////////////
static void FreeResource( void ){
  // checkCudaErrors(hipDeviceSynchronize());
  // glFinish();
  // HANDLE_ERROR( hipGraphicsUnregisterResource(resource1));
  // HANDLE_ERROR( hipGraphicsUnregisterResource(resourceL));
  // checkCudaErrors( hipGraphicsUnregisterResource(resource1) );
  // hipGraphicsUnregisterResource(resource1);
  // hipGraphicsUnregisterResource(resource2);
  // deletePBO(&bufferObj);
  // deletePBO(&bufferObj2);
  // deletePBO(&bufferObjL);
  chemA = 0;
  chemB = 0;
  // deleteTexture(&textureID);
  // glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 );
  // glDeleteTextures(1, &textureID);
  glDeleteBuffers(1, &bufferObj);
  exit(0);
}

///////////////////////////////////////
// Keyboard
///////////////////////////////////////
static void key_func( unsigned char key, int x, int y ) {
  switch (key) {
    case 'q':
        FreeResource();
        break;
    case 32:
        draw_func();
        break;
    case 'p':
        togSimulate = !togSimulate;
        break;
    case '=':
        simulate();
        draw_func();
        break;
    case '.':
        writeCpy = true;
        break;
    default:
        break;
  }
}

void passive(int x1, int y1) {
    mouse_old_x = x1; 
    mouse_old_y = y1;
    // glutPostRedisplay();

}

///////////////////////////////////////////////////////////////////////////////
// Main
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {

  // initialize
  glutInit( &argc, argv );
  glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
  glutInitWindowSize( DIM, DIM );
  glutCreateWindow( "sort test" );
  glewInit();
  checkCudaErrors(cudaGLSetGLDevice( 0 ));


  glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
  displayPtr = (float4*)malloc(sizeof(float4)*DIM*DIM);

  // on create openGL
  glGenBuffers( 1, &bufferObj );
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj );
  glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, sizeof(float4) * DIM * DIM, NULL, GL_DYNAMIC_DRAW_ARB );
  hipGraphicsGLRegisterBuffer( &resource1, bufferObj, cudaGraphicsMapFlagsWriteDiscard );
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 );

  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &textureID);
  glBindTexture(GL_TEXTURE_2D, textureID);
  glTexImage2D( GL_TEXTURE_2D, 0, GL_RGBA, DIM, DIM, 0, GL_BGRA, GL_FLOAT, NULL);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);


// set up GLUT and kick off main loop
  glutCloseFunc( FreeResource );
  glutKeyboardFunc( key_func );
  glutPassiveMotionFunc(passive);
  // glutIdleFunc( simulate );
  glutDisplayFunc( draw_func );
  glutMainLoop();

  hipDeviceReset();

}

