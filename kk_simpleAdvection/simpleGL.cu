#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "writePNG.h"
#include "simpleGL_kernels.cuh"

#define MAX(a,b) ((a > b) ? a : b)
#define SWAP(x0,x) {float *tmp=x0;x0=x;x=tmp;}

dim3 grid, threads;

int size = 0;
int win_x = 512;
int win_y = 512;
float dt = 0.1;
float diff = 0.;
float visc = 0.;
float force = 5.0;
float source_density = 100.0;

GLuint  bufferObj, bufferObj2;
GLuint  textureID;
// hipGraphicsResource_t resource[2];
hipGraphicsResource_t resource1;
hipGraphicsResource_t resource2;
// float ttime = 0.0f;
// float avgFPS = 0.0f;
// int fpsCount = 0;        // FPS count for averaging
// int fpsLimit = 1;        // FPS limit for sampling
int frameNum = 0;
unsigned int frameCount = 0;

float *u, *v, *u_prev, *v_prev, *source, *dens, *dens_prev;
float *dens_cpu;
float4 *displayPtr, *toDisplay;

bool hasRunOnce = false;

// mouse controls
static int mouse_down[3];
int mouse_x, mouse_y, mouse_x_old, mouse_y_old;
bool togSimulate = true;
int max_simulate = 0;

bool writeCpy = false;
bool writeDone = false;


int ID(int i, int j) { return (i+((N+2)*j)); }

///////////////////////////////////////////////////////////////////////////////
// Initialize Variables
///////////////////////////////////////////////////////////////////////////////
void initVariables() {
  grid = dim3(DIM/16,DIM/16);
  threads = dim3(16,16);

  size = (N+2)*(N+2);
  displayPtr = (float4*)malloc(sizeof(float4)*DIM*DIM);
  dens_cpu = (float*)malloc(sizeof(float)*size);
}

///////////////////////////////////////////////////////////////////////////////
// Initialize OpenGL
///////////////////////////////////////////////////////////////////////////////
void initGL(int argc, char *argv[]) {
  glutInit( &argc, argv );
  glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
  glutInitWindowPosition ( 0, 0 );
  // glutInitWindowSize( DIM, DIM );
  glutInitWindowSize ( win_x, win_y );
  glutCreateWindow( "Simple Advection" );
  glewInit();

  glGenBuffers( 1, &bufferObj );
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj );
  glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, sizeof(float4) * DIM * DIM, NULL, GL_DYNAMIC_DRAW_ARB );
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 );

  glEnable(GL_TEXTURE_2D);
  glGenTextures(1, &textureID);
  glBindTexture(GL_TEXTURE_2D, textureID);
  glTexImage2D( GL_TEXTURE_2D, 0, GL_RGBA, DIM, DIM, 0, GL_BGRA, GL_FLOAT, NULL);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

  glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();

}

void initCUDA() {
  checkCudaErrors(cudaGLSetGLDevice( 0 ));
  hipGraphicsGLRegisterBuffer( &resource1, bufferObj, cudaGraphicsMapFlagsWriteDiscard );

  checkCudaErrors(hipMalloc((void**)&u, sizeof(float)*size ));
  checkCudaErrors(hipMalloc((void**)&u_prev, sizeof(float)*size ));
  checkCudaErrors(hipMalloc((void**)&v, sizeof(float)*size ));
  checkCudaErrors(hipMalloc((void**)&v_prev, sizeof(float)*size ));
  checkCudaErrors(hipMalloc((void**)&dens, sizeof(float)*size ));
  checkCudaErrors(hipMalloc((void**)&dens_prev, sizeof(float)*size ));
  checkCudaErrors(hipMalloc((void**)&source, sizeof(float)*size ));
  checkCudaErrors(hipMalloc((void**)&toDisplay, sizeof(float4)*size ));
}

void initArrays() {
  ClearArray<<<grid,threads>>>(u, 0.0);
  ClearArray<<<grid,threads>>>(u_prev, 0.0);
  ClearArray<<<grid,threads>>>(v, 0.0);
  ClearArray<<<grid,threads>>>(v_prev, 0.0);
  ClearArray<<<grid,threads>>>(dens, 0.0);
  ClearArray<<<grid,threads>>>(dens_prev, 0.0);
  ClearArray<<<grid,threads>>>(toDisplay, 0.0);
}

///////////////////////////////////////////////////////////////////////////////
// Sim steps
///////////////////////////////////////////////////////////////////////////////
void get_from_UI(float *d, float *u, float *v) {

  int i, j = (N+2)*(N+2);

  WeirdThing<<<grid,threads>>>(d, u, v); // rename this

  if ( !mouse_down[0] && !mouse_down[2] ) return;

  // map mouse position to window size
  i = (int)((mouse_x /(float)win_x)*N+1);
	j = (int)(((win_y-mouse_y)/(float)win_y)*N+1);

  float x_diff = mouse_x-mouse_x_old;
  float y_diff = mouse_y_old-mouse_y;
  if (frameNum % 50 == 0) printf("%f, %f\n", x_diff, y_diff);

  if ( i<1 || i>N || j<1 || j>N ) return;

  // DrawSquare<<<grid,threads>>>(dens_prev);

  if ( mouse_down[0] ) {
    GetFromUI<<<grid,threads>>>(u, i, j, x_diff * force);
    GetFromUI<<<grid,threads>>>(v, i, j, y_diff * force);
  }

  if ( mouse_down[2]) {
    GetFromUI<<<grid,threads>>>(d, i, j, source_density);
  }

  mouse_x_old = mouse_x;
  mouse_y_old = mouse_y;

  return;
}

void diffuse_step(int b, float *field, float *field0, float diff, float dt){
  float a=dt*diff*float(N)*float(N); // needed to float(N) to get it to work...
  for (int k = 0; k < 20; k++) {
    LinSolve<<<grid,threads>>>( b, field, field0, a, (float)1.0+(4.0*a) );
    SetBoundary<<<grid,threads>>>(0, field);
  }
}

void proj_step( float *u, float *v, float *p, float *div) {
    Project<<<grid,threads>>>( u, v, p, div );
    SetBoundary<<<grid,threads>>>(0, div);
    SetBoundary<<<grid,threads>>>(0, p);
    for (int k = 0; k < 20; k++) {
      LinSolve<<<grid,threads>>>( 0, p, div, 1.0, 4.0 );
      SetBoundary<<<grid,threads>>>(0, p);
    }
    ProjectFinish<<<grid,threads>>>( u, v, p, div );
    SetBoundary<<<grid,threads>>>(1, u);
    SetBoundary<<<grid,threads>>>(2, v);
}

void dens_step ( float * x, float * x0, float * u, float * v, float diff, float dt )
{
  AddSource<<<grid,threads>>>( x, x0, dt );
  SWAP ( x0, x );
  diffuse_step( 0, x, x0, diff, dt);

  SWAP ( x0, x );
  Advect<<<grid,threads>>>( 0, x, x0, u, v, dt );
  SetBoundary<<<grid,threads>>>(0, x);
}

void vel_step ( float * u, float * v, float * u0, float * v0, float visc, float dt ) {
  AddSource<<<grid,threads>>>( u, u0, dt );
  AddSource<<<grid,threads>>>( v, v0, dt );

  SWAP ( u0, u ); diffuse_step( 1, u, u0, visc, dt);
  SWAP ( v0, v ); diffuse_step( 2, v, v0, visc, dt);

  proj_step( u, v, u0, v0);

  SWAP ( u0, u );
  SWAP ( v0, v );
  Advect<<<grid,threads>>>( 1, u, u0, u0, v0, dt ); SetBoundary<<<grid,threads>>>(1, u);
  Advect<<<grid,threads>>>( 2, v, v0, u0, v0, dt ); SetBoundary<<<grid,threads>>>(2, v);

  proj_step( u, v, u0, v0);
}

///////////////////////////////////////////////////////////////////////////////
// Simulate
///////////////////////////////////////////////////////////////////////////////
static void simulate( void ){

  // *!* important
  if (!hasRunOnce) {
    initArrays();
    hasRunOnce = true;
  }

  if (frameNum > 0 && togSimulate) {
    get_from_UI(dens_prev, u_prev, v_prev);
    vel_step( u, v, u_prev, v_prev, visc, dt );
    dens_step( dens, dens_prev, u, v, diff, dt );
    MakeColor<<<grid,threads>>>(dens, toDisplay);
  }

  size_t  sizeT;
  hipGraphicsMapResources( 1, &resource1, 0 );
  checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&displayPtr, &sizeT, resource1));
  checkCudaErrors(hipMemcpy(displayPtr, toDisplay, sizeof(float4)*size, hipMemcpyDeviceToHost ));
  checkCudaErrors(hipGraphicsUnmapResources( 1, &resource1, 0 ));

  checkCudaErrors(hipMemcpy(dens_cpu, dens, sizeof(float)*size, hipMemcpyDeviceToHost ));

  frameNum++;
  glutPostRedisplay();
}



static void pre_display ( void )
{
	glViewport ( 0, 0, win_x, win_y );
	glMatrixMode ( GL_PROJECTION );
	glLoadIdentity ();
	gluOrtho2D ( 0.0, 1.0, 0.0, 1.0 );
	glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
}

///////////////////////////////////////////////////////////////////////////////
// Draw
///////////////////////////////////////////////////////////////////////////////
static void draw_func( void ) {

  glViewport ( 0, 0, win_x, win_y );
  glMatrixMode ( GL_PROJECTION );
  glLoadIdentity ();
  gluOrtho2D ( 0.0, 1.0, 0.0, 1.0 );

  glClear(GL_COLOR_BUFFER_BIT);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
  glBindTexture(GL_TEXTURE_2D, textureID);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, DIM, DIM, GL_BGRA, GL_FLOAT, NULL);

  glBegin(GL_QUADS);
  glTexCoord2f( 0, 1.0f);
  glVertex3f(0.0,1.0,0.0);
  glTexCoord2f(0,0);
  glVertex3f(0.0,0.0,0.0);
  glTexCoord2f(1.0f,0);
  glVertex3f(1.0f,0.0,0.0);
  glTexCoord2f(1.0f,1.0f);
  glVertex3f(1.0,1.0,0.0);
  glEnd();

  // pre_display ();

  // int i, j;
  // float x, y, h, d00, d01, d10, d11;
  //
  // h = 1.0f/float(N);
  //
  // glBegin ( GL_QUADS );
  //   for ( i=0 ; i<=N ; i++ ) {
  //     x = (float)(i-0.5f)*h;
  //     for ( j=0 ; j<=N ; j++ ) {
  //       y = (float)(j-0.5f)*h;
  //
  //       d00 = dens_cpu[ID(i,j)];
  //       d01 = dens_cpu[ID(i,j+1)];
  //       d10 = dens_cpu[ID(i+1,j)];
  //       d11 = dens_cpu[ID(i+1,j+1)];
  //
  //       glColor4f ( d00, d00, d00, 1.0 ); glVertex2f ( x, y );
  //       glColor4f ( d10, d10, d10, 1.0 ); glVertex2f ( x+h, y );
  //       glColor4f ( d11, d11, d11, 1.0 ); glVertex2f ( x+h, y+h );
  //       glColor4f ( d01, d01, d01, 1.0 ); glVertex2f ( x, y+h );
  //     }
  //   }
  // glEnd ();

  glutSwapBuffers();

  // glutPostRedisplay(); // causes draw to loop forever

}

///////////////////////////////////////////////////////////////////////////////
// Misc functions
///////////////////////////////////////
// Write
///////////////////////////////////////
void write(const char* _filename, float4* _img) {
  FILE* file;
  file = fopen(_filename, "wb");

  int totalCells = DIM * DIM;
  for (int i = 0; i < totalCells; i++) {
    fprintf(file, "%f\n", _img[i].x);
    fprintf(file, "%f\n", _img[i].y);
    fprintf(file, "%f\n", _img[i].z);
  }

  fclose(file);

  writeCpy = false;
  printf("Wrote file!\n");
}
///////////////////////////////////////
// Close
///////////////////////////////////////
static void FreeResource( void ){
  // checkCudaErrors(hipFree(u));
  // checkCudaErrors(hipFree(u_prev));
  // checkCudaErrors(hipFree(v));
  // checkCudaErrors(hipFree(v_prev));
  // checkCudaErrors(hipFree(dens));
  // checkCudaErrors(hipFree(dens_prev));
  // checkCudaErrors(hipFree(source));
  // checkCudaErrors(hipFree(toDisplay));
  glDeleteBuffers(1, &bufferObj);
}

///////////////////////////////////////
// Keyboard
///////////////////////////////////////
static void key_func( unsigned char key, int x, int y ) {
  switch (key) {
    case 'q':
    case 'Q':
        FreeResource();
        exit(0);
        break;
    case 32:
        draw_func();
        break;
    case 'p':
        togSimulate = !togSimulate;
        break;
    case '=':
        simulate();
        draw_func();
        break;
    case '.':
        writeCpy = true;
        break;
    case ']':
        diff += .1;
        if (diff >= 1.) diff = 1.;
        printf("Diff: %f\n", diff);
        break;
    case '[':
        diff -= .1;
        if (diff <= 0.) diff = 0.;
        printf("Diff: %f\n", diff);
        break;
    case '0':
        visc += .1;
        if (visc >= 1.) visc = 1.;
        printf("Visc: %f\n", visc);
        break;
    case '9':
        visc -= .1;
        if (visc <= 0.) visc = 0.;
        printf("Visc: %f\n", visc);
        break;
    default:
        break;
  }
}

///////////////////////////////////////////////////////////////////////////////
// GLUT Mouse
///////////////////////////////////////////////////////////////////////////////
void motion_func(int x, int y) {
  mouse_x = x;
  mouse_y = y;
}

void mouse_func ( int button, int state, int x, int y )
{
	mouse_x_old = mouse_x = x;
	mouse_y_old = mouse_x = y;

	mouse_down[button] = state == GLUT_DOWN;
}

static void reshape_func ( int width, int height )
{
	// glutSetWindow ( win_id );
	glutReshapeWindow ( width, height );

	win_x = width;
	win_y = height;
}
///////////////////////////////////////////////////////////////////////////////
// Main
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {

  // initialize
  initVariables();
  initGL(argc, argv);
  initCUDA();

  // pre_display ();


// set up GLUT and kick off main loop
  // glutCloseFunc( FreeResource );
  glutKeyboardFunc( key_func );
  glutMouseFunc ( mouse_func );
  glutMotionFunc(motion_func);
  glutIdleFunc( simulate );
  glutReshapeFunc ( reshape_func );
  glutDisplayFunc( draw_func );
  glutMainLoop();

  hipDeviceReset();

}
